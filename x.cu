#include <hip/hip_runtime.h>
#include <iostream>

__global__ void simpleKernel(int *data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] += 1;  // A simple operation, e.g., increment each element
    }
}

int main() {
    const int size = 1024;
    int *hostData = nullptr, *deviceData = nullptr;

    // Allocate host memory
    hostData = new int[size];

    // Initialize host data
    for (int i = 0; i < size; ++i) {
        hostData[i] = i;
    }

    // Allocate device memory
    hipMalloc(&deviceData, size * sizeof(int));

    // Create two CUDA streams
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // Copy data from host to device asynchronously in stream1
    hipMemcpyAsync(deviceData, hostData, size * sizeof(int), hipMemcpyHostToDevice, stream1);

    // Launch kernel in stream2
    dim3 blockSize(256);
    dim3 gridSize((size + blockSize.x - 1) / blockSize.x);
    simpleKernel<<<gridSize, blockSize, 0, stream2>>>(deviceData, size);

    // Copy data back from device to host asynchronously in stream1
    hipMemcpyAsync(hostData, deviceData, size * sizeof(int), hipMemcpyDeviceToHost, stream1);

    // Synchronize streams
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    // Cleanup
    hipFree(deviceData);
    delete[] hostData;
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    return 0;
}
