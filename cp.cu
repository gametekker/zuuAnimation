#include <iostream>
#include <hip/hip_runtime.h>

// Simple CUDA Kernel for demonstration
__global__ void addKernel(int *d_a, int *d_b, int *d_c, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        d_c[i] = d_a[i] + d_b[i];
    }
}

int main() {
    const int size = 1024;
    int *h_a, *h_b, *h_c; // Host arrays
    int *d_a, *d_b, *d_c; // Device arrays

    // Allocate host memory
    h_a = new int[size];
    h_b = new int[size];
    h_c = new int[size];

    // Initialize host arrays
    for (int i = 0; i < size; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Allocate device memory
    hipMalloc(&d_a, size * sizeof(int));
    hipMalloc(&d_b, size * sizeof(int));
    hipMalloc(&d_c, size * sizeof(int));

    // Create a CUDA stream
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Copy data from host to device asynchronously
    hipMemcpyAsync(d_a, h_a, size * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_b, h_b, size * sizeof(int), hipMemcpyHostToDevice, stream);

    // Launch kernel in the stream
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    addKernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_a, d_b, d_c, size);

    // Copy result back to host asynchronously
    hipMemcpyAsync(h_c, d_c, size * sizeof(int), hipMemcpyDeviceToHost, stream);

    // Synchronize the stream (wait for all operations in the stream to complete)
    hipStreamSynchronize(stream);

    // Host-side processing after synchronization
    std::cout << "Results after kernel execution:" << std::endl;
    for (int i = 0; i < 5; i++) { // Print first 5 results
        std::cout << "h_c[" << i << "] = " << h_c[i] << std::endl;
    }

    // Clean up
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    hipStreamDestroy(stream);

    return 0;
}
